#include <stdio.h>
#include <chrono>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <bits/stdc++.h>
#include <bitset>
#include <fstream>
#include <sstream>
#include <stdlib.h>
using namespace std;
using namespace std::chrono;
#define MAX 18446744073709551615
#define SATCLAUSES 3
/*
    Parameters
*/
#define threadsPerBlock 1024
#define N threadsPerBlock * 216 // perents
#define roundsPerMigration 10
#define threadsInBlockIsland 32                                                        // number of threads in an island
#define totalThreadsIsland N / threadsInBlockIsland                                    // number of islands
#define totalBlocksIsland (totalThreadsIsland + threadsPerBlock - 1) / threadsPerBlock // number of blocks for islands
#define selectionMode true                                                             // elitism or ranking selection
#define crossoverMode 2                                                                //  uniform or single-point or two-points crossover
#define mutationMode false                                                             // single or double mutation
#define mutationKeep false                                                             // exempt the best parent to be mutated
#define mutationThreshold 0.5                                                          // between 0 and 1
#define terminationMode true                                                           // terminate by rounds without improvement or by time
#define maxRound 20000
#define debugMode false // print rounds or print MAXSAT evaluation format

// https://stackoverflow.com/questions/65293876/cuda-gpuassert-an-illegal-memory-access-was-encountered
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__constant__ int d_satSize; // the number of clauses

__constant__ int d_maxBit; // variables (bits) needed for the problem set

__constant__ short d_satSets[10000 * SATCLAUSES]; // problem set literals

// initialise a state with a random state
__global__ void init(hiprandState_t *states, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        hiprand_init(clock64(), id, 0, &states[id]);
    }
}

// initialise a random integer number
__global__ void random_casting_int(hiprandState_t *states, int *numbers, int maxIndex, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        numbers[id] = hiprand_uniform(&states[id]) * maxIndex;
    }
}

// initialise a random float number
__global__ void random_casting_float(hiprandState_t *states, float *numbers, int maxIndex, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        numbers[id] = hiprand_uniform(&states[id]) * maxIndex;
    }
}

// initialise a parent which is of type unsigned long long int
__global__ void random_casting_parent(hiprandState_t *states, unsigned long long int *numbers, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        numbers[id] = hiprand_uniform_double(&states[id]) * MAX;
    }
}

// evaluate the fitness value of a chromosome
__global__ void evaluation(unsigned long long int *parents, unsigned int *parentVals, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int tempVal = 0;
        for (int i = 0; i < d_satSize; i++)
        {
            for (int ii = 0; ii < SATCLAUSES; ii++)
            {
                if ((d_satSets[i * SATCLAUSES + ii] < 0 && (!((parents[id] >> abs(d_satSets[i * SATCLAUSES + ii]) - 1) & 1))) ||
                    (d_satSets[i * SATCLAUSES + ii] > 0 && ((parents[id] >> abs(d_satSets[i * SATCLAUSES + ii]) - 1) & 1)))
                {
                    // if sat sets literal is negative and chromosome index is 0 or sat sets literal is positive and chromosome index is 1
                    tempVal += 1;
                    break;
                }
            }
        }
        parentVals[id] = tempVal;
    }
}

__global__ void mutation(unsigned long long int *parents, float *mutateProb, int *mutateIndex, bool mode, bool isMutationKeep, int bestIndex, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        if (!isMutationKeep || id != bestIndex)
        {
            if (mode)
            {
                // Single Mutation
                if (mutateProb[id] > mutationThreshold)
                {
                    if (!((parents[id] >> mutateIndex[id]) & 1))
                    {
                        // if chromsome idth index is 0
                        parents[id] |= (1ULL << mutateIndex[id]);
                    }
                    else
                    {
                        // if chromsome idth index is 1
                        parents[id] &= ~(1ULL << mutateIndex[id]);
                    }
                }
            }
            else
            {
                // Double Mutation
                if (!((parents[id] >> mutateIndex[id]) & 1))
                {
                    // if chromsome idth index is 0
                    parents[id] |= (1ULL << mutateIndex[id]);
                }
                else
                {
                    // if chromsome idth index is 1
                    parents[id] &= ~(1ULL << mutateIndex[id]);
                }
                int nextId = id + 1;
                if (nextId > N)
                    nextId = nextId - N;
                if (mutateProb[id] > mutationThreshold)
                {
                    if (!((parents[id] >> mutateIndex[nextId]) & 1))
                    {
                        // if chromsome nextIdth index is 0
                        parents[id] |= (1ULL << mutateIndex[nextId]);
                    }
                    else
                    {
                        // if chromsome nextIdth index is 1
                        parents[id] &= ~(1ULL << mutateIndex[nextId]);
                    }
                }
            }
        }
    }
}

// crossover for single-point and two-points crossover
__global__ void crossover_fixed(unsigned long long int *parents, unsigned long long int *blockBestParents, int *splitIndex, int *length, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int startingPosition = splitIndex[id] - length[id];
    if (startingPosition < 0)
        startingPosition = 0;
    if (max > id)
    {
        int bId = blockIdx.x;
        for (int i = startingPosition; i < splitIndex[id]; i++)
        {
            if ((blockBestParents[bId] >> i) & 1)
            {
                // if selected chromsome ith index is 1
                if (!((parents[id] >> i) & 1))
                {
                    // if chromsome ith index is 0
                    parents[id] |= (1ULL << i);
                }
            }
            else
            {
                // if selected chromsome ith index is 0
                if ((parents[id] >> i) & 1)
                {
                    // if chromsome ith index is 1
                    parents[id] &= ~(1ULL << i);
                }
            }
        }
    }
}

__global__ void crossover_uniform(unsigned long long *parents, unsigned long long *blockBestParents, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = blockIdx.x;
        for (int i = 0; i < d_maxBit; i += 2)
        {
            if ((blockBestParents[bId] >> i) & 1)
            {
                // if selected chromsome ith index is 1
                if (!((parents[id] >> i) & 1))
                {
                    // if chromsome ith index is 0
                    parents[id] |= (1ULL << i);
                }
            }
            else
            {
                // if selected chromsome ith index is 0
                if ((parents[id] >> i) & 1)
                {
                    // if chromsome ith index is 1
                    parents[id] &= ~(1ULL << i);
                }
            }
        }
    }
}

__global__ void selection_elitism(unsigned long long int *parents, unsigned int *parentVals, unsigned long long int *blockBestParent, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * threadsInBlockIsland;
        unsigned int tmpLargestVal = 0;
        unsigned long long int tmpLargestPar = 0;
        // iterate over the threads in an island
        for (int i = 0; i < threadsInBlockIsland; i++)
        {
            if (parentVals[bId + i] > tmpLargestVal)
            {
                tmpLargestPar = parents[bId + i];
                tmpLargestVal = parentVals[bId + i];
            }
        }
        // select the chromosome with the highest fitness value at the corresponding blockBestParent array
        blockBestParent[id] = tmpLargestPar;
    }
}

__global__ void selection_wheel(unsigned long long int *parents, unsigned int *parentVals, unsigned long long int *blockBestParent, float *wheelProbs, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * threadsInBlockIsland;
        unsigned int tmpLowestVal = d_satSize + 100;
        unsigned int totalVal = 0;
        // find the lowest and total fitness value
        for (int i = 0; i < threadsInBlockIsland; i++)
        {
            if (parentVals[bId + i] < tmpLowestVal)
            {
                tmpLowestVal = parentVals[bId + i];
            }
            totalVal += parentVals[bId + i];
        }
        unsigned int base = totalVal - threadsInBlockIsland * tmpLowestVal;
        // store the cumulative proabability
        float tmpProb = 0;
        for (int i = 0; i < threadsInBlockIsland; i++)
        {
            tmpProb += (parentVals[bId + i] - tmpLowestVal) / base;
            if (tmpProb > wheelProbs[id])
            {
                // select the chromosome when the probability is higher than the randomly generated probability
                blockBestParent[id] = parents[bId + i];
                break;
            }
        }
    }
}

__global__ void internalReOrder(unsigned long long int *parents, unsigned int *parentVals, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * threadsInBlockIsland;
        int lowestIndex, highestIndex, highestVal = 0;
        int lowestVal = d_satSize + 100;
        ;
        for (int i = 0; i < threadsInBlockIsland; i++)
        {
            // store the chromsomes with the lowest and highest fitness values
            if (i == 0)
            {
                lowestVal = parentVals[bId + i];
                highestVal = parentVals[bId + i];
                lowestIndex = bId + i;
                highestIndex = bId + i;
            }
            else
            {
                if (parentVals[bId + i] < lowestVal)
                {
                    lowestVal = parentVals[bId + i];
                    lowestIndex = bId + i;
                }
                else if (parentVals[bId + i] > highestVal)
                {
                    highestVal = parentVals[bId + i];
                    highestIndex = bId + i;
                }
            }
        }
        unsigned long long int tmpLowest = parents[lowestIndex];
        unsigned long long int tmpHighest = parents[highestIndex];
        // swap the position of the first position with that of the chromosome with lowest fitness values
        parents[lowestIndex] = parents[bId];
        parents[bId] = tmpLowest;
        // swap the position of the last position with that of the chromosome with highest fitness values
        parents[highestIndex] = parents[bId + threadsInBlockIsland - 1];
        parents[bId + threadsInBlockIsland - 1] = tmpHighest;
    }
}

__global__ void migration(unsigned long long int *parents, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        // Migration - the last chromosome replaces the first chromosome of the next block
        int index = (id + 1) * threadsInBlockIsland - 1;
        if (index >= N)
            index = index - N;
        int replaceIndex = (id + 1) * threadsInBlockIsland;
        if (replaceIndex >= N)
            replaceIndex = replaceIndex - N;
        parents[replaceIndex] = parents[index];
    }
}

short *readSatSets(string fileName, int *h_maxBit, int *h_satSize)
{
    string tempText;
    // Read from the text file
    ifstream firstFileRead(fileName);
    ifstream secondFileRead(fileName);
    int tmpSatSize = 0;
    int tmpMaxBit = 0;
    while (getline(firstFileRead, tempText))
    {
        // process each line
        if (tempText[0] == 'p')
        {
            // if the line is about problem definition
            istringstream iss(tempText);
            string s;
            int tmpIndex = 0;
            while (getline(iss, s, ' '))
            {
                // process strings splitted by space in a line
                if (tmpIndex == 2)
                {
                    tmpMaxBit = atoi(s.c_str());
                }
                else if (tmpIndex == 3)
                {
                    tmpSatSize = atoi(s.c_str());
                }
                if (!(tmpIndex >= 2 && atoi(s.c_str()) == 0))
                    tmpIndex += 1;
            }
            break;
        }
    }
    bool isCount = false;
    int index = 0;
    short *h_satSets = new short[tmpSatSize * SATCLAUSES];
    while (getline(secondFileRead, tempText))
    {
        // process each line
        if (tempText[0] == 'p')
        {
            isCount = true;
        }
        else if (isCount && tempText[0] != 'c')
        {
            // iterate over all the literals in a line
            string tmpStr;
            for (int i = 0; i < tempText.size(); i++)
            {

                if (tempText[i] != ' ')
                {
                    tmpStr += tempText[i];
                }
                else
                {
                    if (tmpStr != "0" && !tmpStr.empty())
                    {
                        short tmpNumber = stoi(tmpStr);
                        h_satSets[index] = tmpNumber;
                        tmpStr = "";
                        index += 1;
                    }
                }
            }
        };
    }
    *h_maxBit = tmpMaxBit;
    *h_satSize = tmpSatSize;
    firstFileRead.close();
    secondFileRead.close();
    return h_satSets;
}

// print the bits of a chromosome
void printBits(unsigned long long int parent, int max)
{
    cout << parent << endl;
    for (int i = 0; i < max; i++)
    {
        if ((parent >> i) & 1)
        {
            cout << (i + 1);
        }
        else
        {
            cout << (i + 1) * -1;
        }
        cout << " ";
    }
    cout << endl;
}

void printOccupancyMetrics(int blockSize, int minGridSize, string funcName)
{
    cout << "--------------------------------------" << endl;
    cout << "Function Name: " << funcName << endl;
    cout << "Block Size: " << blockSize << endl;
    cout << "Min Grid Size: " << minGridSize << endl;
}

// print the max block and grid size for all the kernel functions in the program
void printMaxBlockSize()
{
    int blockSize;
    int minGridSize;
    int max_active_blocks;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_sm = 0;
    hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, 0);
    cout << "Number of Streaming Multiprocessor: " << num_sm << endl;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks, init, blockSize, 0);
    cout << "Maximum active blocks:  " << max_active_blocks << endl;
    cout << "--------------------------------------" << endl;
    printOccupancyMetrics(blockSize, minGridSize, "init");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, random_casting_int, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "random_casting_int");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, random_casting_float, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "random_casting_float");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, random_casting_parent, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "random_casting_parent");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, evaluation, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "evaluation");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mutation, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "mutation");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, crossover_fixed, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "crossover_fixed");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, crossover_uniform, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "crossover_uniform");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, selection_elitism, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "selection_elitism");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, selection_wheel, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "selection_wheel");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, internalReOrder, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "internalReOrder");
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, migration, 0, 0);
    printOccupancyMetrics(blockSize, minGridSize, "migration");
    cout << "--------------------------------------" << endl;
}

int main(int argc, char **argv)
{
    // printMaxBlockSize();
    srand(time(0));
    auto start = high_resolution_clock::now();
    string fileName = "";
    if (argc == 2)
        fileName = argv[1];
    else if (argc > 2)
        std::invalid_argument("too many arguments");
    else
        throw std::invalid_argument("no file specified");
    if (!debugMode)
    {
        cout << "c ------------------------------------" << endl;
        cout << "c CUDA Genetic Algorithm MAXSAT solver" << endl;
        cout << "c ------------------------------------" << endl;
    }

    int h_maxBit = 0;
    int h_satSize = 0;
    short *h_satSets = readSatSets(fileName, &h_maxBit, &h_satSize);
    hipMemcpyToSymbol(HIP_SYMBOL(d_maxBit), &h_maxBit, 1 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_satSize), &h_satSize, 1 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_satSets), h_satSets, h_satSize * SATCLAUSES * sizeof(short));

    hiprandState_t *d_parent_states;
    gpuErrchk(hipMalloc((void **)&d_parent_states, N * sizeof(hiprandState_t)));
    unsigned long long int *d_parents;
    gpuErrchk(hipMalloc(&d_parents, N * sizeof(unsigned long long int)));

    // initialise states for parents
    init<<<N / threadsPerBlock, threadsPerBlock>>>(d_parent_states, N);
    gpuErrchk(hipPeekAtLastError());

    // initialise values for parents
    random_casting_parent<<<N / threadsPerBlock, threadsPerBlock>>>(d_parent_states, d_parents, N);
    gpuErrchk(hipPeekAtLastError());

    // initialise an array to store fitness values of parents in device
    unsigned int *d_parentVals;
    gpuErrchk(hipMalloc(&d_parentVals, N * sizeof(unsigned int)));

     // initialise arrays to store fitness values of parents in host
    unsigned long long int *h_parents;
    h_parents = (unsigned long long int *)malloc(N * sizeof(unsigned long long int));

    // initialise an array to store fitness values of parents in device
    unsigned int *h_parentVals;
    h_parentVals = (unsigned int *)malloc(N * sizeof(unsigned int));
    unsigned long long int *d_block_bests;
    gpuErrchk(hipMalloc(&d_block_bests, totalBlocksIsland * sizeof(unsigned long long int)));

    // initialise all the device arrays
    hiprandState_t *d_crossover_states, *d_crossover_length_states, *d_mutation_index_states, *d_mutation_prob_status, *d_selection_prob_status;
    int *d_crossover_index, *d_crossover_length_index, *d_mutation_index;
    float *d_selection_prob, *d_mutation_prob;
    // crossover index
    gpuErrchk(hipMalloc((void **)&d_crossover_states, N * sizeof(hiprandState_t)));
    gpuErrchk(hipMalloc(&d_crossover_index, N * sizeof(int)));
    // crossover length
    gpuErrchk(hipMalloc((void **)&d_crossover_length_states, N * sizeof(hiprandState_t)));
    gpuErrchk(hipMalloc(&d_crossover_length_index, N * sizeof(int)));
    // mutation index
    gpuErrchk(hipMalloc((void **)&d_mutation_index_states, N * sizeof(hiprandState_t)));
    gpuErrchk(hipMalloc(&d_mutation_index, N * sizeof(int)));
    // mutation probability
    gpuErrchk(hipMalloc((void **)&d_mutation_prob_status, N * sizeof(hiprandState_t)));
    gpuErrchk(hipMalloc(&d_mutation_prob, N * sizeof(float)));
    // selection probability
    gpuErrchk(hipMalloc((void **)&d_selection_prob_status, totalThreadsIsland * sizeof(hiprandState_t)));
    gpuErrchk(hipMalloc(&d_selection_prob, N * sizeof(float)));

    // variables
    unsigned int res_maxParentVal = 0;
    unsigned long long int res_maxParent = 0;
    auto res_maxTime = start;
    int roundsWithoutImprovement = 0;
    int res_maxRound = 0;
    int roundIndex = 0;

    // while loop starts here
    while ((terminationMode && (roundsWithoutImprovement < maxRound)))
    {
        if (roundIndex % roundsPerMigration == (roundsPerMigration / 2))
        {
            // Migration
            if (debugMode)
                cout << "<- Migration ->" << endl;
            internalReOrder<<<totalBlocksIsland, threadsPerBlock>>>(d_parents, d_parentVals, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
            evaluation<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_parentVals, N);
            gpuErrchk(hipPeekAtLastError());
            /* migration */
            migration<<<totalBlocksIsland, threadsPerBlock>>>(d_parents, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
        }
        evaluation<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_parentVals, N);
        gpuErrchk(hipPeekAtLastError());
        hipMemcpy(h_parentVals, d_parentVals, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(h_parents, d_parents, N * sizeof(unsigned long long int), hipMemcpyDeviceToHost);

        // selection
        if (selectionMode)
        {
            // Elitism selection
            selection_elitism<<<totalBlocksIsland, threadsPerBlock>>>(d_parents, d_parentVals, d_block_bests, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
        }
        else
        {
            // Roulette Wheel Selection
            init<<<totalBlocksIsland, threadsPerBlock>>>(d_selection_prob_status, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
            random_casting_float<<<totalBlocksIsland, threadsPerBlock>>>(d_selection_prob_status, d_selection_prob, 1, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
            selection_wheel<<<totalBlocksIsland, threadsPerBlock>>>(d_parents, d_parentVals, d_block_bests, d_selection_prob, totalThreadsIsland);
            gpuErrchk(hipPeekAtLastError());
        }
        gpuErrchk(hipPeekAtLastError());

        // Find the highest fitness value in that round
        int tempLargestParentIndex = 0;
        unsigned int tempLargestParentValue = 0;
        evaluation<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_parentVals, N);
        gpuErrchk(hipPeekAtLastError());
        hipMemcpy(h_parentVals, d_parentVals, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(h_parents, d_parents, N * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
        for (int i = 0; i < N; i++)
        {
            // If the chromsome with the highest fitness value in that round is found, store the value
            if (h_parentVals[i] > tempLargestParentValue)
            {
                tempLargestParentValue = h_parentVals[i];
                tempLargestParentIndex = i;
            }
        }

        if (debugMode)
        {
            // If it is in debug mode, print the highest fitness value each round
            cout << roundIndex << ". "
                 << "answer: ";
            cout << tempLargestParentValue << "  parent: ";
            cout << h_parents[tempLargestParentIndex] << " ";
            cout << "at " << tempLargestParentIndex << endl;
        }

        if (tempLargestParentValue > res_maxParentVal)
        {
            // If the chromsome with the highest fitness value so far is found, end the program
            res_maxParentVal = tempLargestParentValue;
            res_maxParent = h_parents[tempLargestParentIndex];
            res_maxRound = roundIndex;
            res_maxTime = high_resolution_clock::now();
            roundsWithoutImprovement = -1;
            if (!debugMode)
            {
                if (h_satSize == res_maxParentVal)
                {
                    // If all the clauses are satisfied
                    cout << "s OPTIMUM FOUND" << endl;
                    cout << "v ";
                    printBits(res_maxParent, h_maxBit);
                    cout << endl;
                    break;
                }
                else
                {
                    cout << "o " << (h_satSize - res_maxParentVal) << endl;
                }
            }
        }

        // crossover
        if (crossoverMode)
        {
            // single-point crossover
            init<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_states, N);
            gpuErrchk(hipPeekAtLastError());
            random_casting_int<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_states, d_crossover_index, h_maxBit, N);
            gpuErrchk(hipPeekAtLastError());
            crossover_fixed<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_block_bests, d_crossover_index, d_crossover_index, N);
        }
        else if (crossoverMode == 2)
        {
            // two-point crossover
            init<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_states, N);
            gpuErrchk(hipPeekAtLastError());
            random_casting_int<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_states, d_crossover_index, h_maxBit, N);
            gpuErrchk(hipPeekAtLastError());
            init<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_length_states, N);
            gpuErrchk(hipPeekAtLastError());
            random_casting_int<<<N / threadsPerBlock, threadsPerBlock>>>(d_crossover_length_states, d_crossover_length_index, h_maxBit, N);
            gpuErrchk(hipPeekAtLastError());
            crossover_fixed<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_block_bests, d_crossover_index, d_crossover_length_index, N);
        }
        else
        {
            // uniform crossover
            crossover_uniform<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_block_bests, N);
        }
        gpuErrchk(hipPeekAtLastError());

        // mutation
        // initialise new random states and numbers for mutation
        init<<<N / threadsPerBlock, threadsPerBlock>>>(d_mutation_index_states, N);
        gpuErrchk(hipPeekAtLastError());
        random_casting_int<<<N / threadsPerBlock, threadsPerBlock>>>(d_mutation_index_states, d_mutation_index, h_maxBit, N);
        gpuErrchk(hipPeekAtLastError());
        init<<<N / threadsPerBlock, threadsPerBlock>>>(d_mutation_prob_status, N);
        gpuErrchk(hipPeekAtLastError());
        random_casting_float<<<N / threadsPerBlock, threadsPerBlock>>>(d_mutation_prob_status, d_mutation_prob, 1, N);
        gpuErrchk(hipPeekAtLastError());
        mutation<<<N / threadsPerBlock, threadsPerBlock>>>(d_parents, d_mutation_prob, d_mutation_index, mutationMode, mutationKeep, tempLargestParentIndex, N);
        gpuErrchk(hipPeekAtLastError());

        roundIndex += 1;
        roundsWithoutImprovement += 1;
    }
    /*
        Print Final Result
    */
    if (debugMode)
    {
        cout << "-------------LARGEST PARENT-------------" << endl;
        cout << "Number of bits : " << h_maxBit << endl;
        cout << "SAT Size: " << h_satSize << endl;
        cout << "Best parent: " << res_maxParent << endl;
        cout << "Best parent: " << bitset<64>(res_maxParent).to_string() << endl;
        cout << "Best value: " << res_maxParentVal << endl;
        cout << "Best round: " << res_maxRound << endl;
        auto duration_max = duration_cast<microseconds>(res_maxTime - start);
        cout << "Best round time: " << duration_max.count() << endl;
        cout << "Total round: " << roundIndex << endl;
        cout << "----------------------------------" << endl;
        cout << "Selection Mode: ";
        if (selectionMode)
            cout << "Elitism" << endl;
        else
            cout << "Roulette Wheel" << endl;
        cout << "Crossover Mode: ";
        if (crossoverMode==1)
            cout << "Single Point" << endl;
        else if(crossoverMode==2)
            cout << "Two-Points" << endl;
        else
            cout << "Uniform" << endl;
        cout << "Mutation Mode: ";
        if (mutationMode)
            cout << "Single" << endl;
        else
            cout << "Double" << endl;
        cout << "Termination Mode: ";
        if (terminationMode)
            cout << "Rounds" << endl;
        else
            cout << "Time" << endl;
    }

    /* Free device memory */
    hipFree(d_parents);
    hipFree(d_block_bests);
    hipFree(d_parentVals);
    hipFree(d_parent_states);
    hipFree(d_selection_prob_status);
    hipFree(d_crossover_states);
    hipFree(d_crossover_length_states);
    hipFree(d_mutation_index_states);
    hipFree(d_mutation_prob_status);
    hipFree(d_selection_prob);
    hipFree(d_crossover_index);
    hipFree(d_crossover_length_index);
    hipFree(d_mutation_index);
    hipFree(d_mutation_prob);

    /* Free host memory */
    free(h_parents);
    free(h_parentVals);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: "
         << duration.count() << " microseconds" << endl;
    if (h_satSize == res_maxParentVal)
        return 1;
    else
        return 0;
}